#include "hip/hip_runtime.h"

#include "cuda_TD1.hpp"

#include <hip/hip_runtime.h>

#include <iostream>

namespace {

__global__ void nx2_plus_ny(int n, int m, int* TabX, int* TabY, int* TabOut, int TailleTab)
{
	int idGlobal =
		threadIdx.x // <= 0 - 511
		+ blockIdx.x * blockDim.x // x * 512
		+ blockIdx.y * blockDim.x * gridDim.x; // y * 512 * 1000

	if (idGlobal < TailleTab)
		TabOut[idGlobal] = n * TabX[idGlobal] * TabX[idGlobal] + m * TabY[idGlobal];
}

} // namespace

void nx2_plus_ny_GPU()
{
    int n;
	int m;
	std::cin >> n;
	std::cin >> m;
    int TailleTab;
	std::cin >> TailleTab;
	fdim Grille(1000, 1000); // <= D�finir la taille de la grille en fonction de la taille du tableau
	int* TabX = new int[TailleTab];
	int* TabY = new int[TailleTab];
	int* TabOut = new int[TailleTab];
	// random fill tabX and tabY
	for (int i = 0; i < TailleTab; i++)
	{
		TabX[i] = rand() % 100;
		TabY[i] = rand() % 100;
	}
	nx2_plus_ny_GPU << <Grille, 512 >> > (n, m, TabX, TabY, TabOut, TailleTab);
	delete[] TabX;
	delete[] TabY;
	delete[] TabOut;

	auto err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Error on runOnGPU: " << hipGetErrorString(err) << std::endl;
	}
}
